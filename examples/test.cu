#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void testkernel(__device__ float* a, __device__ float* b, __device__ float* c)
{
	unsigned int id = threadIdx.x;
	c[id] = a[id] + b[id];
}

void print_cuda_devices()
{
	int num = 0;
  	struct hipDeviceProp_t devprop;

  	hipGetDeviceCount(&num);
  	if(!num)
    {
    	std::cout << "No CUDA devices found." << std::endl;
      	return;
    }

  	for(int i = 0; i < num; i++)
	{
  		hipGetDeviceProperties(&devprop, i);
      	std::cout << "Found CUDA device: " << devprop.name << std::endl;
		std::cout << "\tCompute Capability: " << devprop.major << "." << devprop.minor << std::endl;
		std::cout << "\tMultiprocessor Count: " << devprop.multiProcessorCount << std::endl;
		std::cout << "\tClock Rate: " << static_cast<float>(devprop.clockRate)/1024.0f/1024.0f << "Ghz" << std::endl;
		std::cout << "\tTotal Global: " << devprop.totalGlobalMem/1024 << "MB" << std::endl;
		std::cout << "\tTotal L2 Cache: " << devprop.l2CacheSize << "KB" << std::endl;
	}
}

#define TESTSIZE 32
int main(int argc, char* argv[])
{
	print_cuda_devices();

	float *da, *db, *dc, *a, *b, *c;

	size_t pitch;

   	hipMallocPitch((void**) &da, &pitch, sizeof(float), TESTSIZE);
   	hipMallocPitch((void**) &db, &pitch, sizeof(float), TESTSIZE);
   	hipMallocPitch((void**) &dc, &pitch, sizeof(float), TESTSIZE);

   	a = new float[TESTSIZE];
   	b = new float[TESTSIZE];
   	c = new float[TESTSIZE];

   	for(int i = 0; i < TESTSIZE; i++) a[i] = i;
	for(int i = 0; i < TESTSIZE; i++) b[i] = i;

   	hipMemcpy2D(da, pitch, a, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyHostToDevice);
	hipMemcpy2D(db, pitch, b, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyHostToDevice);

	testkernel<<<1, TESTSIZE>>>(da, db, dc);

	hipMemcpy2D(dc, pitch, c, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyDeviceToHost);

	for(int i = 0; i < TESTSIZE; i++)
		printf("%f + %f = %f\n", a[i], b[i], c[i]);
}
