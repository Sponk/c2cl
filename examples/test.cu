#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void testkernel(__device__ float* a, __device__ float* b, __device__ float* c)
{
	unsigned int id = threadIdx.x;
	c[id] = a[id] + b[id];
}

#define TESTSIZE 32
int main(int argc, char* argv[])
{
	float *da, *db, *dc, *a, *b, *c;

	size_t pitch;

   	hipMallocPitch((void**) &da, &pitch, sizeof(float), TESTSIZE);
   	hipMallocPitch((void**) &db, &pitch, sizeof(float), TESTSIZE);
   	hipMallocPitch((void**) &dc, &pitch, sizeof(float), TESTSIZE);

   	a = new float[TESTSIZE];
   	b = new float[TESTSIZE];
   	c = new float[TESTSIZE];

   	for(int i = 0; i < TESTSIZE; i++) a[i] = i;
	for(int i = 0; i < TESTSIZE; i++) b[i] = i;

   	hipMemcpy2D(da, pitch, a, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyHostToDevice);
	hipMemcpy2D(db, pitch, b, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyHostToDevice);

	testkernel<<<1, TESTSIZE>>>(da, db, dc);

	hipMemcpy2D(dc, pitch, c, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyDeviceToHost);

	for(int i = 0; i < TESTSIZE; i++)
		printf("%f + %f = %f\n", a[i], b[i], c[i]);
}
