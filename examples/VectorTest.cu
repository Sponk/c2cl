#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include "common.h"

__global__ void add(float* a, float* b, float* c)
{
	unsigned int id = threadIdx.x;
	c[id] = a[id] + b[id];
}

__global__ void mul(float* a, float* b, float* c)
{
	unsigned int id = threadIdx.x;
	c[id] = a[id] * b[id];
}

__global__ void div(float* a, float* b, float* c)
{
	unsigned int id = threadIdx.x;
	c[id] = a[id] / b[id];
}

#define TESTSIZE 32
int main(int argc, char* argv[])
{
	float *da, *db, *dc, *a, *b, *c;

	size_t pitch;

	CUDA_CHECK(hipMalloc((void**) &da, sizeof(float) * TESTSIZE));
	CUDA_CHECK(hipMalloc((void**) &db, sizeof(float) * TESTSIZE));
	CUDA_CHECK(hipMalloc((void**) &dc, sizeof(float) * TESTSIZE));

   	a = new float[TESTSIZE];
   	b = new float[TESTSIZE];
   	c = new float[TESTSIZE];

   	for(int i = 0; i < TESTSIZE; i++) a[i] = i;
	for(int i = 0; i < TESTSIZE; i++) b[i] = TESTSIZE - i;

	CUDA_CHECK(hipMemcpy(da, a, sizeof(float) * TESTSIZE, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(db, b, sizeof(float) * TESTSIZE, hipMemcpyHostToDevice));

	// Test addition kernel
	add<<<1, TESTSIZE>>>(da, db, dc);
	CUDA_CHECK_LAST;

	CUDA_CHECK(hipMemcpy2D(dc, pitch, c, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyDeviceToHost));

	// Check expected result
	int retval = 0;
	for(int i = 0; i < TESTSIZE; i++)
		if(fabs(c[i] - (a[i] + b[i])) > 0.0001)
		{
			std::cerr << "Error: Result is not as expected!" << std::endl;
			retval = 1;
		}

	CUDA_CHECK(hipFree(da));
	CUDA_CHECK(hipFree(db));
	CUDA_CHECK(hipFree(dc));

	delete[] a;
	delete[] b;
	delete[] c;

	return retval;
}