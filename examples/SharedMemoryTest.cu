#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include "common.h"

__global__ void countThreads(int* output)
{
	__shared__ int counter;
	counter = 0;

	__syncthreads();
	atomicInc(&counter, 0);
	__syncthreads();

	if (threadIdx.x == 0)
		*output = counter;
}

int main(int argc, char* argv[])
{
	int* output;
	hipMallocManaged((void**) &output, 4);
	*output = 0;
	countThreads<<<1, 32>>>(output);
	hipDeviceSynchronize();

	if(*output != 32)
	{
		std::cerr << *output << std::endl;
		hipFree(output);
		return 1;
	}

	hipFree(output);
	return 0;
}
