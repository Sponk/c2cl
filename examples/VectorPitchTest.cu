#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include "common.h"

__global__ void add(float* a, float* b, float* c)
{
	unsigned int id = threadIdx.x;
	c[id] = a[id] + b[id];
}

__global__ void mul(float* a, float* b, float* c)
{
	unsigned int id = threadIdx.x;
	c[id] = a[id] * b[id];
}

__global__ void div(float* a, float* b, float* c)
{
	unsigned int id = threadIdx.x;
	c[id] = a[id] / b[id];
}

#define TESTSIZE 32
int main(int argc, char* argv[])
{
	float *da, *db, *dc, *a, *b, *c;

	size_t pitch;

   	CUDA_CHECK(hipMallocPitch((void**) &da, &pitch, sizeof(float), TESTSIZE));
   	CUDA_CHECK(hipMallocPitch((void**) &db, &pitch, sizeof(float), TESTSIZE));
   	CUDA_CHECK(hipMallocPitch((void**) &dc, &pitch, sizeof(float), TESTSIZE));

   	a = new float[TESTSIZE];
   	b = new float[TESTSIZE];
   	c = new float[TESTSIZE];

   	for(int i = 0; i < TESTSIZE; i++) a[i] = i;
	for(int i = 0; i < TESTSIZE; i++) b[i] = TESTSIZE - i;

   	CUDA_CHECK(hipMemcpy2D(da, pitch, a, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy2D(db, pitch, b, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyHostToDevice));

	// Test addition kernel
	add<<<1, 32>>>(da, db, dc);
	CUDA_CHECK_LAST;

	CUDA_CHECK(hipMemcpy2D(dc, pitch, c, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyDeviceToHost));

	// Check expected result
	int retval = 0;
	for(int i = 0; i < TESTSIZE; i++)
		if(fabs(c[i] - (a[i] + b[i])) > 0.0001)
		{
			std::cerr << "Error: Result is not as expected!" << std::endl;
			retval = 1;
		}

	CUDA_CHECK(hipFree(da));
	CUDA_CHECK(hipFree(db));
	CUDA_CHECK(hipFree(dc));

	delete[] a;
	delete[] b;
	delete[] c;

	return retval;
}