#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include "common.h"

__global__ void addAbs(float* a, float* b, float* c)
{
	unsigned int id = threadIdx.x;
	c[id] = fabs(a[id] + b[id]);
}

#define TESTSIZE 32
int main(int argc, char* argv[])
{
	float *da, *db, *dc, *a, *b, *c;

	size_t pitch;

	CUDA_CHECK(hipMalloc((void**) &da, sizeof(float) * TESTSIZE));
	CUDA_CHECK(hipMalloc((void**) &db, sizeof(float) * TESTSIZE));
	CUDA_CHECK(hipMalloc((void**) &dc, sizeof(float) * TESTSIZE));

   	a = new float[TESTSIZE];
   	b = new float[TESTSIZE];
   	c = new float[TESTSIZE];

   	for(int i = 0; i < TESTSIZE; i++) a[i] = i;
	for(int i = 0; i < TESTSIZE; i++) b[i] = TESTSIZE - i;

	CUDA_CHECK(hipMemcpy(da, a, sizeof(float) * TESTSIZE, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(db, b, sizeof(float) * TESTSIZE, hipMemcpyHostToDevice));

	// Test addition kernel
	addAbs<<<1, 32>>>(da, db, dc);
	CUDA_CHECK_LAST;

	CUDA_CHECK(hipMemcpy2D(dc, pitch, c, sizeof(float), sizeof(float), TESTSIZE, hipMemcpyDeviceToHost));

	// Check expected result
	int retval = 0;
	for(int i = 0; i < TESTSIZE; i++)
		if(fabs(c[i] - fabs(a[i] + b[i])) > 0.0001)
		{
			std::cerr << "Error: Result is not as expected!" << std::endl;
			retval = 1;
		}

	CUDA_CHECK(hipFree(da));
	CUDA_CHECK(hipFree(db));
	CUDA_CHECK(hipFree(dc));

	delete[] a;
	delete[] b;
	delete[] c;

	return retval;
}
